#include "hip/hip_runtime.h"
#include "LinearLayer.hpp"
#include "hip/hip_runtime.h"
#include ""
#include <stdexcept>
// #include <iostream>

#define TILE_SIZE 32

__global__ void linear_layer_kernal(
    float* weight_matrix, float* biases, 
    float* x_inputs, float* z_values,
    int num_output_neurons, int num_input_neurons,
    int batch_size
)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < num_output_neurons * batch_size)
    {
        int batch_index = index / num_output_neurons;
        int neuron_index = index % num_output_neurons;

        z_values[index] = biases[neuron_index];
        for (int i = 0; i < num_input_neurons; i++)
        {
            z_values[index] += weight_matrix[neuron_index * num_input_neurons + i] * 
                               x_inputs[batch_index * num_input_neurons + i];
        }
    }
}

__global__ void transpose(float *input, float *output, int width, int height) {
    __shared__ float tile[TILE_SIZE][TILE_SIZE + 1]; // +1 to avoid bank conflicts

    int x = blockIdx.x * TILE_SIZE + threadIdx.x;
    int y = blockIdx.y * TILE_SIZE + threadIdx.y;

    // Load data into shared memory
    if (x < width && y < height) {
        tile[threadIdx.y][threadIdx.x] = input[y * width + x];
    }

    __syncthreads();

    // Transpose and write back to global memory
    x = blockIdx.y * TILE_SIZE + threadIdx.x;
    y = blockIdx.x * TILE_SIZE + threadIdx.y;

    if (x < height && y < width) {
        output[y * height + x] = tile[threadIdx.x][threadIdx.y];
    }
}

__global__ void linear_layer_backward_weights_kernal(
    float* weights_gradient, float* output_gradient,
    float* prev_input, int num_output_neurons, 
    int num_input_neurons, int batch_size
)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < num_output_neurons * num_input_neurons) {
        int output_idx = index / num_input_neurons;
        int input_idx = index % num_input_neurons;

        for (int i = 0; i < batch_size; i++) {
            weights_gradient[index] += output_gradient[i * num_output_neurons + output_idx] * prev_input[i * num_input_neurons + input_idx];
        }
        weights_gradient[index] /= batch_size;
    }
}

__global__ void linear_layer_backward_inputs_kernal(
    float* input_gradient, float* output_gradient,
    float* weights_transposed, int num_output_neurons, 
    int num_input_neurons, int batch_size
)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < num_input_neurons * batch_size) {
        int batch_index = index / num_input_neurons;
        int input_index = index % num_input_neurons;

        input_gradient[index] = 0;

        for (int i = 0; i < num_output_neurons; i++) {
            input_gradient[index] += weights_transposed[input_index * num_output_neurons + i] * output_gradient[batch_index * num_output_neurons + i];
        }
    }
}

__global__ void linear_layer_backward_biases_kernal(
    float* biases_gradient, float* output_gradient,
    int num_output_neurons, int batch_size
)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < num_output_neurons) {
        for (int i = 0; i < batch_size; i++) {
            biases_gradient[index] += output_gradient[i * num_output_neurons + index];
        }
        biases_gradient[index] /= batch_size;
    }
}

void LinearLayer::forward(float* input, int batch_size) {
    // if the arrays are not allocated/defined, raise an error
    if (weights == nullptr || biases == nullptr || z_values == nullptr || activations == nullptr) {
        throw std::runtime_error("Arrays are not allocated/defined");
    }
    // prev_input = input;
    hipMemcpy(prev_input, input, num_inputs * sizeof(float) * batch_size, hipMemcpyDeviceToDevice);
    linear_layer_kernal<<< (num_outputs * batch_size + 255) / 256, 256 >>>(weights, biases, input, z_values, num_outputs, num_inputs, batch_size);
    hipDeviceSynchronize();
}

void LinearLayer::backward(float* output_gradient, float* input_gradient, float* weights_gradient, float* biases_gradient, int batch_size) {
    // if the arrays are not allocated/defined, raise an error
    if (weights_gradient == nullptr || biases_gradient == nullptr || input_gradient == nullptr) {
        throw std::runtime_error("Arrays are not allocated/defined");
    }

    int threads_per_block = 256;
    int blocks = (num_outputs + threads_per_block - 1) / threads_per_block;

    linear_layer_backward_biases_kernal<<<blocks, threads_per_block>>>(biases_gradient, output_gradient, num_outputs, batch_size);
    hipDeviceSynchronize();
    // error = hipGetLastError();
    // if (error != hipSuccess) {
    //     throw std::runtime_error("Error in backward biases kernel: " + std::string(hipGetErrorString(error)));
    // }


    // bias gradient is just the output gradient
    // hipError_t error = hipMemcpy(biases_gradient, output_gradient, num_outputs * sizeof(float), hipMemcpyDeviceToDevice); // this only works for single item
    
    // TODO: add batching current implementation only works for single item

    // if (error != hipSuccess) {
    //     throw std::runtime_error("Error copying biases gradient: " + std::string(hipGetErrorString(error)));
    // }

    // transpose weights
    float* weights_transposed;
    error = hipMalloc(&weights_transposed, num_outputs * num_inputs * sizeof(float));
    // if (error != hipSuccess) {
    //     throw std::runtime_error("Error allocating weights_transposed: " + std::string(hipGetErrorString(error)));
    // }

    // got the transpose kernel from somewhere online
    dim3 blockSize(TILE_SIZE, TILE_SIZE);
    dim3 gridSize((num_inputs + TILE_SIZE - 1) / TILE_SIZE, (num_outputs + TILE_SIZE - 1) / TILE_SIZE);
    transpose<<<gridSize, blockSize>>>(weights, weights_transposed, num_inputs, num_outputs);
    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != hipSuccess) {
        hipFree(weights_transposed);
        throw std::runtime_error("Error in transpose kernel: " + std::string(hipGetErrorString(error)));
    }

    // Print prev_input values
    float* host_prev_input = new float[num_inputs];
    hipMemcpy(host_prev_input, prev_input, num_inputs * sizeof(float), hipMemcpyDeviceToHost);
    // std::cout << "prev_input: ";
    // for (int i = 0; i < num_inputs; i++) {
    //     std::cout << host_prev_input[i] << ", ";
    // }
    // std::cout << std::endl;
    // delete[] host_prev_input;

    blocks = (num_outputs * num_inputs + threads_per_block - 1) / threads_per_block;

    linear_layer_backward_weights_kernal<<<blocks, threads_per_block>>>(weights_gradient, output_gradient, prev_input, num_outputs, num_inputs, batch_size);
    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != hipSuccess) {
        hipFree(weights_transposed);
        throw std::runtime_error("Error in backward weights kernel: " + std::string(hipGetErrorString(error)));
    }

    blocks = (num_inputs * batch_size + threads_per_block - 1) / threads_per_block;

    linear_layer_backward_inputs_kernal<<<blocks, threads_per_block>>>(input_gradient, output_gradient, weights_transposed, num_outputs, num_inputs, batch_size);
    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != hipSuccess) {
        hipFree(weights_transposed);
        throw std::runtime_error("Error in backward inputs kernel: " + std::string(hipGetErrorString(error)));
    }

    error = hipFree(weights_transposed);
    if (error != hipSuccess) {
        throw std::runtime_error("Error freeing weights_transposed: " + std::string(hipGetErrorString(error)));
    }
}
