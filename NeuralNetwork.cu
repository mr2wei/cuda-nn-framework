#include "hip/hip_runtime.h"
#include "NeuralNetwork.hpp"
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <string>
#include <fstream>
#include <sstream>
#include <vector>
#include <cmath>
#include <random>
#include <iostream>

// Helper function to initialize common parts of the constructors
void NeuralNetwork::initialize(std::vector<NNLayer*> layers, float* host_weights, float* host_biases) {
    total_weights = 0;
    total_b_z_a = 0;
    total_input_gradient = 0;
    
    for (NNLayer* layer : layers) {
        if (!layer->is_activation_layer) {
            shape.push_back(layer->num_inputs);
        }
        total_input_gradient += layer->num_inputs;
    }
    shape.push_back(layers.back()->num_outputs);

    
    for (int i = 0; i < shape.size() - 1; i++) {
        total_weights += shape[i] * shape[i + 1];
        total_b_z_a += shape[i + 1];
    }


    bool rand_weights_biases = false;

    if (!host_weights && !host_biases) {
        // Create random weights and biases
        host_weights = new float[total_weights];
        host_biases = new float[total_b_z_a];

        int offset = 0;
        for (int i = 0; i < shape.size() - 1; i++) {
            glorot_uniform_weights(shape[i], shape[i + 1], host_weights, offset);
            offset += shape[i] * shape[i + 1];
        }
        
        for (int i = 0; i < total_b_z_a; i++) {
            host_biases[i] = 0;
        }
        rand_weights_biases = true;
    }

    // Allocate memory for weights, biases, z_values, and activations
    hipMalloc(&device_weights, total_weights * sizeof(float));
    hipMalloc(&device_biases, total_b_z_a * sizeof(float));
    hipMalloc(&device_z_values, total_b_z_a * sizeof(float));
    hipMalloc(&device_activations, total_b_z_a * sizeof(float));

    // Copy weights, biases, z_values, and activations to device if provided
    if (host_weights && host_biases) {
        hipMemcpy(device_weights, host_weights, total_weights * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(device_biases, host_biases, total_b_z_a * sizeof(float), hipMemcpyHostToDevice);
    }

    // Initialize layers
    size_t weights_offset = 0, b_z_a_offset = 0, input_gradient_offset = 0;
    for (int i = 0; i < layers.size(); i++) {
        // if the first layer is an activation layer, raise an error
        if (i == 0 && layers[i]->is_activation_layer) {
            throw std::runtime_error("First layer cannot be an activation layer");
        }

        // Check for consecutive activation layers
        if (i > 0 && layers[i]->is_activation_layer && layers[i-1]->is_activation_layer) {
            throw std::runtime_error("Cannot have consecutive activation layers");
        }

        layers[i]->weights = device_weights + weights_offset;
        layers[i]->biases = device_biases + b_z_a_offset;
        layers[i]->z_values = device_z_values + b_z_a_offset;
        layers[i]->activations = device_activations + b_z_a_offset;

        hipMalloc(&layers[i]->prev_input, layers[i]->num_inputs * sizeof(float));

        if (!layers[i]->is_activation_layer) {
            weights_offset += layers[i]->num_inputs * layers[i]->num_outputs;
        }

        // Only increment offsets if the next layer is not an activation layer
        bool next_layer_is_activation = (i < layers.size() - 1) && layers[i + 1]->is_activation_layer;
        if (!next_layer_is_activation) {
            b_z_a_offset += layers[i]->num_outputs;
        }
    }

    if (rand_weights_biases) {
        delete[] host_weights;
        delete[] host_biases;
    }
}

void NeuralNetwork::glorot_uniform_weights(int input_size, int output_size, float* host_weights, int offset) {
    float limit = std::sqrt(6.0f / (input_size + output_size));
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis(-limit, limit);
    for (int i = 0; i < input_size * output_size; i++) {
        host_weights[i + offset] = dis(gen);
    }
}

NeuralNetwork::NeuralNetwork(std::vector<NNLayer*> layers)
    : layers(layers) {
    initialize(layers, nullptr, nullptr);
}

NeuralNetwork::NeuralNetwork(std::vector<NNLayer*> layers, float* host_weights, float* host_biases)
    : layers(layers) {
    initialize(layers, host_weights, host_biases);
}

NeuralNetwork::~NeuralNetwork() {
    hipFree(device_weights);
    hipFree(device_biases);
    hipFree(device_z_values);
    hipFree(device_activations);
}

/**
 * @brief Forward pass through the neural network
 * 
 * @param input: input to the neural network
 */
void NeuralNetwork::forward(float* input) {
    // reset activations and z_values
    hipMemset(device_activations, 0, total_b_z_a * sizeof(float));
    hipMemset(device_z_values, 0, total_b_z_a * sizeof(float));

    // Allocate device memory for input
    float* device_input;
    hipMalloc(&device_input, layers[0]->num_inputs * sizeof(float));
    hipMemcpy(device_input, input, layers[0]->num_inputs * sizeof(float), hipMemcpyHostToDevice);
    
    float* current_input = device_input;

    for (int i = 0; i < layers.size(); i++) {
        NNLayer* layer = layers[i];
        if (layer->is_activation_layer) {
            // For activation layers, the z_values are the input
            layer->forward(layer->z_values);
        } else {
            layer->forward(current_input);
        }
        current_input = layer->activations;
    }

    hipFree(device_input);
}

std::vector<float> NeuralNetwork::get_activations() {
    float* activations_host = new float[total_b_z_a];
    hipError_t err = hipMemcpy(activations_host, device_activations, total_b_z_a * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        delete[] activations_host;
        throw std::runtime_error("Failed to copy activations from device: " + std::string(hipGetErrorString(err)));
    }
    
    std::vector<float> result(activations_host, activations_host + total_b_z_a);
    delete[] activations_host;  // Clean up the temporary array
    return result;
}

std::vector<float> NeuralNetwork::get_z_values() {
    float* z_values_host = new float[total_b_z_a];
    hipError_t err = hipMemcpy(z_values_host, device_z_values, total_b_z_a * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        delete[] z_values_host;
        throw std::runtime_error("Failed to copy z_values from device: " + std::string(hipGetErrorString(err)));
    }
    
    std::vector<float> result(z_values_host, z_values_host + total_b_z_a);
    delete[] z_values_host;
    return result;
}

std::vector<float> NeuralNetwork::get_weights() {
    float* weights_host = new float[total_weights];
    hipError_t err = hipMemcpy(weights_host, device_weights, total_weights * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        delete[] weights_host;
        throw std::runtime_error("Failed to copy weights from device: " + std::string(hipGetErrorString(err)));
    }
    
    std::vector<float> result(weights_host, weights_host + total_weights);
    delete[] weights_host;
    return result;
}

std::vector<float> NeuralNetwork::get_biases() {
    float* biases_host = new float[total_b_z_a];
    hipError_t err = hipMemcpy(biases_host, device_biases, total_b_z_a * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        delete[] biases_host;
        throw std::runtime_error("Failed to copy biases from device: " + std::string(hipGetErrorString(err)));
    }
    
    std::vector<float> result(biases_host, biases_host + total_b_z_a);
    delete[] biases_host;
    return result;
}


std::vector<float> NeuralNetwork::get_results() {
    // Get all values from device
    std::vector<float> activations = get_activations();
    std::vector<float> z_values = get_z_values();
    
    // Get number of outputs from last layer
    int num_outputs = layers.back()->num_outputs;
    int offset = total_b_z_a - num_outputs;

    // Create result vector with proper size
    std::vector<float> result(num_outputs);

    // If last layer is activation layer, return last activations
    // Otherwise return last z_values
    if (layers.back()->is_activation_layer) {
        std::copy(activations.begin() + offset, activations.end(), result.begin());
    } else {
        std::copy(z_values.begin() + offset, z_values.end(), result.begin());
    }

    return result;
}

float NeuralNetwork::get_loss(std::vector<float> target) {
    std::vector<float> results = get_results();
    float loss = 0;
    for (int i = 0; i < results.size(); i++) {
        loss += pow(results[i] - target[i], 2);
    }
    return loss / results.size();
}

float NeuralNetwork::get_loss(float target) {
    if (layers.back()->num_outputs != 1) {
        throw std::runtime_error("Only one target value given but last layer has multiple outputs");
    }
    std::vector<float> results = get_results();
    return pow(results[0] - target, 2);
}

void NeuralNetwork::save_model(std::string filename) {
    std::ofstream file(filename);

    // first line is the shape
    // second line is the order of the layers L = Linear, R = ReLU, S = Sigmoid
    // third line is the weights
    // fourth line is the biases

    for (int shape : shape) {
        file << shape << " ";
    }
    file << std::endl;

    for (int i = 0; i < layers.size(); i++) {
        file << layers[i]->type << " ";
    }
    file << std::endl;

    std::vector<float> weights = get_weights();
    for (float weight : weights) {
        file << weight << " ";
    }
    file << std::endl;

    std::vector<float> biases = get_biases();
    for (float bias : biases) {
        file << bias << " ";
    }
    file << std::endl;

    file.close();
}
