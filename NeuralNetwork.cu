#include "hip/hip_runtime.h"
#include "NeuralNetwork.hpp"
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <string>
#include <fstream>
#include <sstream>
#include <vector>
#include <cmath>
#include <random>
#include <iostream>

// Helper function to initialize common parts of the constructors
void NeuralNetwork::initialize(std::vector<NNLayer*> layers, float* host_weights, float* host_biases) {
    for (NNLayer* layer : layers) {
        if (!layer->is_activation_layer) {
            shape.push_back(layer->num_inputs);
        }
    }
    shape.push_back(layers.back()->num_outputs);

    total_weights = 0;
    total_b_z_a = 0;
    total_input_gradient = 0;
    for (int i = 0; i < shape.size() - 1; i++) {
        total_weights += shape[i] * shape[i + 1];
        total_b_z_a += shape[i + 1];
        total_input_gradient += shape[i];
    }

    total_input_gradient *= 2;

    bool rand_weights_biases = false;

    if (!host_weights && !host_biases) {
        // Create random weights and biases
        host_weights = new float[total_weights];
        host_biases = new float[total_b_z_a];

        int offset = 0;
        for (int i = 0; i < shape.size() - 1; i++) {
            glorot_uniform_weights(shape[i], shape[i + 1], host_weights, offset);
            offset += shape[i] * shape[i + 1];
        }
        
        for (int i = 0; i < total_b_z_a; i++) {
            host_biases[i] = 0;
        }
        rand_weights_biases = true;
    }

    // Allocate memory for weights, biases, z_values, and activations
    hipMalloc(&device_weights, total_weights * sizeof(float));
    hipMalloc(&device_biases, total_b_z_a * sizeof(float));
    hipMalloc(&device_z_values, total_b_z_a * sizeof(float));
    hipMalloc(&device_activations, total_b_z_a * sizeof(float));
    hipMalloc(&device_weights_gradient, total_weights * sizeof(float));
    hipMalloc(&device_biases_gradient, total_b_z_a * sizeof(float));
    hipMalloc(&device_input_gradient, total_input_gradient * sizeof(float));

    // Copy weights, biases, z_values, and activations to device if provided
    if (host_weights && host_biases) {
        hipMemcpy(device_weights, host_weights, total_weights * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(device_biases, host_biases, total_b_z_a * sizeof(float), hipMemcpyHostToDevice);
    }

    // Initialize layers
    size_t weights_offset = 0, b_z_a_offset = 0, input_gradient_offset = 0;
    for (int i = 0; i < layers.size(); i++) {
        // if the first layer is an activation layer, raise an error
        if (i == 0 && layers[i]->is_activation_layer) {
            throw std::runtime_error("First layer cannot be an activation layer");
        }

        // Check for consecutive activation layers
        if (i > 0 && layers[i]->is_activation_layer && layers[i-1]->is_activation_layer) {
            throw std::runtime_error("Cannot have consecutive activation layers");
        }

        layers[i]->weights = device_weights + weights_offset;
        layers[i]->biases = device_biases + b_z_a_offset;
        layers[i]->z_values = device_z_values + b_z_a_offset;
        layers[i]->activations = device_activations + b_z_a_offset;
        layers[i]->input_gradient = device_input_gradient + input_gradient_offset;
        layers[i]->weights_gradient = device_weights_gradient + weights_offset;
        layers[i]->biases_gradient = device_biases_gradient + b_z_a_offset;

        if (!layers[i]->is_activation_layer) {
            weights_offset += layers[i]->num_inputs * layers[i]->num_outputs;
        }

        // Only increment offsets if the next layer is not an activation layer
        bool next_layer_is_activation = (i < layers.size() - 1) && layers[i + 1]->is_activation_layer;
        if (!next_layer_is_activation) {
            b_z_a_offset += layers[i]->num_outputs;
        }

        input_gradient_offset += layers[i]->num_inputs;
    }

    if (rand_weights_biases) {
        delete[] host_weights;
        delete[] host_biases;
    }
}

void NeuralNetwork::glorot_uniform_weights(int input_size, int output_size, float* host_weights, int offset) {
    float limit = std::sqrt(6.0f / (input_size + output_size));
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis(-limit, limit);
    for (int i = 0; i < input_size * output_size; i++) {
        host_weights[i + offset] = dis(gen);
    }
}

NeuralNetwork::NeuralNetwork(std::vector<NNLayer*> layers)
    : layers(layers) {
    initialize(layers, nullptr, nullptr);
}

NeuralNetwork::NeuralNetwork(std::vector<NNLayer*> layers, float* host_weights, float* host_biases)
    : layers(layers) {
    initialize(layers, host_weights, host_biases);
}

NeuralNetwork::~NeuralNetwork() {
    hipFree(device_weights);
    hipFree(device_biases);
    hipFree(device_z_values);
    hipFree(device_activations);
}

/**
 * @brief Forward pass through the neural network
 * 
 * @param input: input to the neural network
 */
void NeuralNetwork::forward(float* input) {
    // reset activations and z_values
    hipMemset(device_activations, 0, total_b_z_a * sizeof(float));
    hipMemset(device_z_values, 0, total_b_z_a * sizeof(float));

    // Allocate device memory for input
    float* device_input;
    hipMalloc(&device_input, layers[0]->num_inputs * sizeof(float));
    hipMemcpy(device_input, input, layers[0]->num_inputs * sizeof(float), hipMemcpyHostToDevice);
    
    float* current_input = device_input;

    for (int i = 0; i < layers.size(); i++) {
        NNLayer* layer = layers[i];
        if (layer->is_activation_layer) {
            // For activation layers, the z_values are the input
            layer->forward(layer->z_values);
        } else {
            layer->forward(current_input);
        }
        current_input = layer->activations;
    }

    hipFree(device_input);
}

void NeuralNetwork::backward(std::vector<float> target) {
    // calculate derivative of loss function
    std::vector<float> results = get_results();
    float* loss_gradient = new float[layers.back()->num_outputs];
    for (int i = 0; i < layers.back()->num_outputs; i++) {
        loss_gradient[i] = (2.0f / results.size()) * (results[i] - target[i]);
    }

    float* device_loss_gradient;
    hipError_t error = hipMalloc(&device_loss_gradient, layers.back()->num_outputs * sizeof(float));
    if (error != hipSuccess) {
        delete[] loss_gradient;
        throw std::runtime_error("Error allocating device_loss_gradient: " + std::string(hipGetErrorString(error)));
    }

    error = hipMemcpy(device_loss_gradient, loss_gradient, layers.back()->num_outputs * sizeof(float), hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        hipFree(device_loss_gradient);
        delete[] loss_gradient;
        throw std::runtime_error("Error copying loss gradient to device: " + std::string(hipGetErrorString(error)));
    }

    float* current_loss_gradient = device_loss_gradient;

    // backward pass through the network
    for (int i = layers.size() - 1; i >= 0; i--) {
        layers[i]->backward(current_loss_gradient);
        error = hipGetLastError();
        if (error != hipSuccess) {
            hipFree(device_loss_gradient);
            delete[] loss_gradient;
            throw std::runtime_error("Error in backward pass for layer " + std::to_string(i) + ": " + std::string(hipGetErrorString(error)));
        }
        current_loss_gradient = layers[i]->input_gradient;
    }

    error = hipFree(device_loss_gradient);
    if (error != hipSuccess) {
        delete[] loss_gradient;
        throw std::runtime_error("Error freeing device_loss_gradient: " + std::string(hipGetErrorString(error)));
    }
    delete[] loss_gradient;
}

void NeuralNetwork::backward(float target) {
    std::vector<float> target_vector(1, target);
    backward(target_vector);
}

void NeuralNetwork::step(float learning_rate) {
    for (int i = 0; i < layers.size(); i++) {
        layers[i]->step(learning_rate);
    }
}

void NeuralNetwork::zero_gradients() {
    hipError_t error = hipMemset(device_weights_gradient, 0, total_weights * sizeof(float));
    if (error != hipSuccess) {
        throw std::runtime_error("Error zeroing weights gradient: " + std::string(hipGetErrorString(error)));
    }

    error = hipMemset(device_biases_gradient, 0, total_b_z_a * sizeof(float));
    if (error != hipSuccess) {
        throw std::runtime_error("Error zeroing biases gradient: " + std::string(hipGetErrorString(error)));
    }

    error = hipMemset(device_input_gradient, 0, total_input_gradient * sizeof(float));
    if (error != hipSuccess) {
        throw std::runtime_error("Error zeroing input gradient: " + std::string(hipGetErrorString(error)));
    }
}

std::vector<float> NeuralNetwork::get_activations() {
    float* activations_host = new float[total_b_z_a];
    hipError_t err = hipMemcpy(activations_host, device_activations, total_b_z_a * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        delete[] activations_host;
        throw std::runtime_error("Failed to copy activations from device: " + std::string(hipGetErrorString(err)));
    }
    
    std::vector<float> result(activations_host, activations_host + total_b_z_a);
    delete[] activations_host;  // Clean up the temporary array
    return result;
}

std::vector<float> NeuralNetwork::get_z_values() {
    float* z_values_host = new float[total_b_z_a];
    hipError_t err = hipMemcpy(z_values_host, device_z_values, total_b_z_a * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        delete[] z_values_host;
        throw std::runtime_error("Failed to copy z_values from device: " + std::string(hipGetErrorString(err)));
    }
    
    std::vector<float> result(z_values_host, z_values_host + total_b_z_a);
    delete[] z_values_host;
    return result;
}

std::vector<float> NeuralNetwork::get_weights() {
    float* weights_host = new float[total_weights];
    hipError_t err = hipMemcpy(weights_host, device_weights, total_weights * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        delete[] weights_host;
        throw std::runtime_error("Failed to copy weights from device: " + std::string(hipGetErrorString(err)));
    }
    
    std::vector<float> result(weights_host, weights_host + total_weights);
    delete[] weights_host;
    return result;
}

std::vector<float> NeuralNetwork::get_biases() {
    float* biases_host = new float[total_b_z_a];
    hipError_t err = hipMemcpy(biases_host, device_biases, total_b_z_a * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        delete[] biases_host;
        throw std::runtime_error("Failed to copy biases from device: " + std::string(hipGetErrorString(err)));
    }
    
    std::vector<float> result(biases_host, biases_host + total_b_z_a);
    delete[] biases_host;
    return result;
}

std::vector<float> NeuralNetwork::get_results() {
    // Get all values from device
    std::vector<float> activations = get_activations();
    std::vector<float> z_values = get_z_values();
    
    // Get number of outputs from last layer
    int num_outputs = layers.back()->num_outputs;
    int offset = total_b_z_a - num_outputs;

    // Create result vector with proper size
    std::vector<float> result(num_outputs);

    // If last layer is activation layer, return last activations
    // Otherwise return last z_values
    if (layers.back()->is_activation_layer) {
        std::copy(activations.begin() + offset, activations.end(), result.begin());
    } else {
        std::copy(z_values.begin() + offset, z_values.end(), result.begin());
    }

    return result;
}

float NeuralNetwork::get_loss(std::vector<float> target) {
    std::vector<float> results = get_results();
    float loss = 0;
    for (int i = 0; i < results.size(); i++) {
        loss += pow(results[i] - target[i], 2);
    }
    return loss / results.size();
}

float NeuralNetwork::get_loss(float target) {
    if (layers.back()->num_outputs != 1) {
        throw std::runtime_error("Only one target value given but last layer has multiple outputs");
    }
    std::vector<float> results = get_results();
    return pow(results[0] - target, 2);
}

void NeuralNetwork::save_model(std::string filename) {
    std::ofstream file(filename);

    // first line is the shape
    // second line is the order of the layers L = Linear, R = ReLU, S = Sigmoid
    // third line is the weights
    // fourth line is the biases

    for (int shape : shape) {
        file << shape << " ";
    }
    file << std::endl;

    for (int i = 0; i < layers.size(); i++) {
        file << layers[i]->type << " ";
    }
    file << std::endl;

    std::vector<float> weights = get_weights();
    for (float weight : weights) {
        file << weight << " ";
    }
    file << std::endl;

    std::vector<float> biases = get_biases();
    for (float bias : biases) {
        file << bias << " ";
    }
    file << std::endl;

    file.close();
}
